#include "hip/hip_runtime.h"
/*─────────────────────────────────────────────────────────────────────────────┐
│       ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔══════╗       │
│       ║ c ║   ║ a ║   ║ n ║   ║ o ║   ║ S ║   ║ i ║   ║ m ║   ║ +  + ║       │
│       ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚══════╝       │
│                                                                              │
│                       ~ Computational Fluid Dynamics ~                       │
│                       High-Performance GPU Flow Solver                       │
│                                                                              │
│                                                                              │
│                                            Copyright (c) 2025 Dário Canossi  │
└─────────────────────────────────────────────────────────────────────────────*/

#include <algorithm>
#include <filesystem>
#include <fstream>

#include "EulerSolver.h"
#include "solverKernels.h"

std::string EulerSolver::controlFile_ = "controls.txt";

EulerSolver::EulerSolver(cartMesh&& mesh)
:
    mesh_(std::make_unique<cartMesh>(std::forward<cartMesh>(mesh))),
    nx_(mesh_->getBlockX() + 2),
    ny_(mesh_->getBlockY() + 2),
    nz_(mesh_->getBlockZ() + 2),
    dx_(mesh_->getLengthX()),
    dy_(mesh_->getLengthY()),
    dz_(mesh_->getLengthZ()),
    totalCells_(nx_ * ny_ * nz_),
    startTime_(-1),
    nIter_(-1),
    CFL_(-1),
    dt_(0.0f),
    totalTime_(0.0f),
    d_U_(nullptr),
    d_residual_(nullptr),
    h_U_(nullptr)
{
    // Read simulation controls
    readControls();

    // Allocate host memory
    h_U_ = new conservativeVars[totalCells_];

    // Allocate device memory
    hipMalloc(&d_U_, totalCells_ * sizeof(conservativeVars));
    hipMalloc(&d_residual_, totalCells_ * sizeof(conservativeVars));

    // Initialise to zero
    hipMemset(d_U_, 0, totalCells_ * sizeof(conservativeVars));
    hipMemset(d_residual_, 0, totalCells_ * sizeof(conservativeVars));
}

EulerSolver::~EulerSolver()
{
    delete[] h_U_;
    hipFree(d_U_);
    hipFree(d_residual_);
}

void EulerSolver::readControls()
{
    std::string inputFile = "./" + controlFile_;
    std::cout << "\nReading control parameters from input file: "
        << inputFile << std::endl;

    // Lambda for removing specific delimiters from input
    auto isDelim = [](const char& chari)
    {
        switch (chari)
        {
            case ' ':
                return true;
            case ';':
                return true;
            default:
                return false;
        }
    };

    // List of all input keywords from the control file
    std::set<std::string> inputKeys
        {
            "startTime",
            "nIter",
            "CFL"
        };

    std::unordered_map<std::string, float> keyFound;

    std::ifstream file(controlFile_);
    std::string str;
    while (std::getline(file, str))
    {
        for (const auto& key : inputKeys)
        {
            if (keyFound.find(key) != keyFound.end()) continue;

            if (str.find(key) != std::string::npos)
            {
                std::string value(str.substr(str.find(key) + key.length()));

                // Remove delimiters from input
                value.erase
                (
                    std::remove_if(value.begin(), value.end(), isDelim),
                    value.end()
                );

                keyFound.insert({key, std::atof(value.c_str())});
            }
        }
    }

    // Sanity check for missing inputs
    if (keyFound.size() != inputKeys.size())
    {
        std::cerr << "Error: Invalid simulation control input!" << std::endl;
        std::exit(1);
    }

    // Construct control data from stream
    startTime_ = keyFound["startTime"];
    nIter_ = keyFound["nIter"];
    CFL_ = keyFound["CFL"];
}

void EulerSolver::solve()
{
    totalTime_ += startTime_;
    std::cout << std::endl;

    // Initialise with uniform flow
    initialiseUniform(1.225f, 50.0f, 10.0f, 0.0f, 101325.0f);

    // Run for the prescribed iterations
    for (int iter = 1; iter <= nIter_; ++iter)
    {
        runTimeStep();

        if (iter % 10 == 0)
        {
            std::cout << "Iteration " << iter << " | dt = "
                << getTimeStep() << "s | Time = " << totalTime_ << std::endl;
        }
    }

    writeSolution("finalSolution.vtk", nIter_);
}

void EulerSolver::initialiseUniform
(
    float rho,
    float u,
    float v,
    float w,
    float p
)
{
    primitiveVars prim(rho, u, v, w, p);
    conservativeVars cons = solverKernels::primitiveToConservative(prim);

    for (size_t i = 0; i < totalCells_; ++i)
    {
        h_U_[i] = cons;
    }

    hipMemcpy
    (
        d_U_,
        h_U_,
        totalCells_ * sizeof(conservativeVars),
        hipMemcpyHostToDevice
    );
}

void EulerSolver::computeResiduals()
{
    dim3 blockSize(8, 8, 4);
    dim3 gridSize
    (
        (nx_ + blockSize.x - 1) / blockSize.x,
        (ny_ + blockSize.y - 1) / blockSize.y,
        (nz_ + blockSize.z - 1) / blockSize.z
    );

    solverKernels::computeResidual<<<gridSize, blockSize>>>
        (d_U_, d_residual_, nx_, ny_, nz_, dx_, dy_, dz_);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error in computeResiduals: "
            << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();
}

float EulerSolver::computeTimeStep()
{
    // For now, use a conservative estimate for max speed of wave propagation.
    // In practice, you'd compute this quantity from the solution.
    float maxWaveSpeed = 100.0f;  // m/s

    float dtx = CFL_ * dx_ / maxWaveSpeed;
    float dty = CFL_ * dy_ / maxWaveSpeed;
    float dtz = CFL_ * dz_ / maxWaveSpeed;

    dt_ = std::min({dtx, dty, dtz});

    // Update total simulation time
    totalTime_ += dt_;

    return dt_;
}

void EulerSolver::applyBoundaryConditions()
{
    int blockSize = 256;
    int gridSize = (totalCells_ + blockSize - 1) / blockSize;

    solverKernels::applyBoundaryConditions<<<gridSize, blockSize>>>
        (d_U_, nx_, ny_, nz_);

    hipDeviceSynchronize();
}

void EulerSolver::updateSolution()
{
    dim3 blockSize(8, 8, 4);
    dim3 gridSize
    (
        (nx_ + blockSize.x - 1) / blockSize.x,
        (ny_ + blockSize.y - 1) / blockSize.y,
        (nz_ + blockSize.z - 1) / blockSize.z
    );

    solverKernels::integrateTime<<<gridSize, blockSize>>>
        (d_U_, d_residual_, dt_, nx_, ny_, nz_);

    hipDeviceSynchronize();
}

void EulerSolver::runTimeStep()
{
    computeTimeStep();
    computeResiduals();
    updateSolution();
    applyBoundaryConditions();
}

void EulerSolver::getSolution(conservativeVars* hostData)
{
    hipMemcpy
    (
        hostData,
        d_U_,
        totalCells_ * sizeof(conservativeVars),
        hipMemcpyDeviceToHost
    );
}

void EulerSolver::writeSolution(const std::string& filename, int iteration)
{
    getSolution(h_U_);

    // Convert to primitive variables for output
    std::vector<float> rho(totalCells_);
    std::vector<float> u(totalCells_);
    std::vector<float> v(totalCells_);
    std::vector<float> w(totalCells_);
    std::vector<float> p(totalCells_);

    for (size_t i = 0; i < totalCells_; ++i)
    {
        primitiveVars prim = solverKernels::conservativeToPrimitive(h_U_[i]);

        rho[i] = prim.rho;
        u[i] = prim.u;
        v[i] = prim.v;
        w[i] = prim.w;
        p[i] = prim.p;
    }

    // Write solution fields to a VTK output file
    std::filesystem::create_directory("solution");
    std::ofstream file("solution/" + filename);

    file << "# vtk DataFile Version 3.0\n";
    file << "Euler Solution at iteration " << iteration << "\n";
    file << "ASCII\n";
    file << "DATASET STRUCTURED_POINTS\n";
    file << "DIMENSIONS " << nx_ << " " << ny_ << " " << nz_ << "\n";
    file << "ORIGIN 0 0 0\n";
    file << "SPACING " << dx_ << " " << dy_ << " " << dz_ << "\n";
    file << "POINT_DATA " << totalCells_ << "\n";

    file << "SCALARS density float 1\n";
    file << "LOOKUP_TABLE default\n";
    for (const auto& val : rho) file << val << "\n";

    file << "SCALARS pressure float 1\n";
    file << "LOOKUP_TABLE default\n";
    for (const auto& val : p) file << val << "\n";

    file << "VECTORS velocity float\n";
    for (size_t i = 0; i < totalCells_; ++i)
    {
        file << u[i] << " " << v[i] << " " << w[i] << "\n";
    }

    file.close();

    std::cout << "\nFields solution written to " << filename << std::endl;
}