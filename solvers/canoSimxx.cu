#include "hip/hip_runtime.h"
/*─────────────────────────────────────────────────────────────────────────────┐
│       ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔══════╗       │
│       ║ c ║   ║ a ║   ║ n ║   ║ o ║   ║ S ║   ║ i ║   ║ m ║   ║ +  + ║       │
│       ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚══════╝       │
│                                                                              │
│                       ~ Computational Fluid Dynamics ~                       │
│                       High-Performance GPU Flow Solver                       │
│                                                                              │
│                                                                              │
│                                            Copyright (c) 2025 Dário Canossi  │
└─────────────────────────────────────────────────────────────────────────────*/

#include <iostream>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include "gFunctions.h"
#include "clock.h"
#include "cartMesh.h"
#include "EulerSolver.h"

void printHeader()
{
    std::cout
        << R"(┌──────────────────────────────────────────────────────────────────────────────┐
│       ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔═══╗   ╔══════╗       │
│       ║ c ║   ║ a ║   ║ n ║   ║ o ║   ║ S ║   ║ i ║   ║ m ║   ║ +  + ║       │
│       ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚═══╝   ╚══════╝       │
│                                                                              │
│                       ~ Computational Fluid Dynamics ~                       │
│                       High-Performance GPU Flow Solver                       │
│                                                                              │
│                                                                              │
│                                            Copyright (c) 2025 Dário Canossi  │
└──────────────────────────────────────────────────────────────────────────────┘
)";
}

void printRunInfo()
{
    // Host name
    char hostBuf[32];
    gethostname(hostBuf, sizeof(hostBuf));

    // Check CUDA availability and abort if no device is found
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)
    {
        std::cerr << "No CUDA devices found! Terminating run." << std::endl;
        std::exit(1);
    }

    // Device ID and properties
    int devID;
    hipDeviceProp_t props;
    hipGetDevice(&devID);
    hipGetDeviceProperties(&props, devID);

    std::cout << "| PID    : " << getpid()
        << "\n| Host   : " << hostBuf
        << "\n| Device : " << "[" << devID << "] " << props.name << " (CC: "
            << props.major << "." << props.minor << ")"
        << "\n| Memory : " << props.totalGlobalMem/(1024*1024) << " Mb"
        << "\n| Date   : " << clock::date().c_str()
        << "\n| Time   : " << clock::clockTime().c_str() << "\n"
        << "########################################"
        << "########################################"
        << "\n" << std::endl;
}

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -*/

int main(int argc, char** argv)
{
    std::unordered_set<std::string> runOpt = {"-mesh", "-init", "-solve"};

    // Parse run option
    if (argc != 2 || !runOpt.count(argv[1]))
    {
        std::cout << "Invalid usage!\nPlease provide one of the following "
            << "run options:\n"
            << "  -mesh\n"
            << "  -init\n"
            << "  -solve\n";
        std::exit(1);
    }

    printHeader();
    printRunInfo();

    std::string runFlag = argv[1];
    runFlag.erase(0, 1);
    if (runFlag == "mesh")
    {
        std::cout << "Creating mesh for simulation..." << std::endl;
        const cartMesh mesh;

        std::cout << "\nEnd." << std::endl;
    }
    else if (runFlag == "init")
    {
        std::cout << "Initialising fields\n" << std::endl;

        std::cout << "End." << std::endl;
    }
    else if (runFlag == "solve")
    {
        std::cout << "Starting simulation\n" << std::endl;

        // Read mesh
        // cartMesh mesh = cartMesh::readMesh();
        cartMesh mesh;

        // Solve Euler equation for inviscid fluid flow
        EulerSolver(std::move(mesh)).solve();

        std::cout << "\nEnd." << std::endl;
    }

    return 0;
}